#include <c10/cuda/CUDAFunctions.h>
#include <hip/hip_runtime.h>
#include <torch/torch.h>
#include <hip/hip_runtime.h>

#ifndef TRANSFORM_CARTESIAN_CUDA_H
#define TRANSFORM_CARTESIAN_CUDA_H
#include "kernel.cu"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), "#x must be on CUDA")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), "#x must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

static const dim3 BLOCKS(const unsigned int threads, const unsigned int len){
    const dim3 blocks( (len + threads -1) / threads ); 
    return blocks; 
}

static const dim3 BLOCKS(
    const unsigned int threads, const unsigned int len, 
    const unsigned int dy,      const unsigned int dz
){
    const dim3 blocks( (len + threads -1) / threads, dy, dz); 
    return blocks; 
}

torch::TensorOptions _MakeOp(torch::Tensor v1){
    return torch::TensorOptions().dtype(v1.scalar_type()).device(v1.device()); 
}

torch::Tensor _Px(torch::Tensor pt, torch::Tensor phi){
    const auto current_device = c10::cuda::current_device();
    c10::cuda::set_device(pt.get_device()); 

    pt = pt.view({-1, 1}).contiguous(); 
    phi = phi.view({-1, 1}).contiguous();
    CHECK_INPUT(pt); CHECK_INPUT(phi);  
    const torch::TensorOptions op = _MakeOp(pt); 

    torch::Tensor px = torch::zeros_like(pt, op);
    const unsigned int threads = 1024;   
    const unsigned int len = pt.size(0); 
    const dim3 blk = BLOCKS(threads, len); 
    
    AT_DISPATCH_FLOATING_TYPES(pt.scalar_type(), "PxK", ([&]
    {
        PxK<scalar_t><<<blk, threads>>>(
            pt.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
           phi.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            px.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            len
        );
    })); 

    c10::cuda::set_device(current_device);
    return px;  
}

torch::Tensor _Py(torch::Tensor pt, torch::Tensor phi){
    const auto current_device = c10::cuda::current_device();
    c10::cuda::set_device(pt.get_device()); 

    pt = pt.view({-1, 1}).contiguous(); 
    phi = phi.view({-1, 1}).contiguous(); 
    CHECK_INPUT(pt); CHECK_INPUT(phi);  
    const torch::TensorOptions op = _MakeOp(pt); 

    torch::Tensor py = torch::zeros_like(pt, op);
    const unsigned int threads = 1024;   
    const unsigned int len = pt.size(0); 
    const dim3 blk = BLOCKS(threads, len); 
    
    AT_DISPATCH_FLOATING_TYPES(pt.scalar_type(), "PyK", ([&]
    {
        PyK<scalar_t><<<blk, threads>>>(
            pt.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
           phi.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            py.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            len
        );
    })); 

    c10::cuda::set_device(current_device);
    return py;  
}

torch::Tensor _Pz(torch::Tensor pt, torch::Tensor eta){
    const auto current_device = c10::cuda::current_device();
    c10::cuda::set_device(pt.get_device()); 

    pt = pt.view({-1, 1}).contiguous(); 
    eta = eta.view({-1, 1}).contiguous(); 
    CHECK_INPUT(pt); CHECK_INPUT(eta);  
    const torch::TensorOptions op = _MakeOp(pt); 

    torch::Tensor pz = torch::zeros_like(pt, op);
    const unsigned int threads = 1024;   
    const unsigned int len = pt.size(0); 
    const dim3 blk = BLOCKS(threads, len); 
    
    AT_DISPATCH_FLOATING_TYPES(pt.scalar_type(), "PzK", ([&]
    {
        PzK<scalar_t><<<blk, threads>>>(
            pt.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
           eta.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            pz.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            len
        );
    })); 

    c10::cuda::set_device(current_device);
    return pz;  
}

torch::Tensor _PxPyPz(torch::Tensor pt, torch::Tensor eta, torch::Tensor phi){
    const auto current_device = c10::cuda::current_device();
    c10::cuda::set_device(pt.get_device()); 

    pt  = pt.view({-1, 1}).contiguous(); 
    eta = eta.view({-1, 1}).contiguous(); 
    phi = phi.view({-1, 1}).contiguous();     
    const torch::TensorOptions op = _MakeOp(pt); 

    torch::Tensor out = torch::zeros_like(pt, op);
    out = torch::cat({out, out, out}, -1).contiguous(); 
    CHECK_INPUT(pt); CHECK_INPUT(eta); CHECK_INPUT(phi); CHECK_INPUT(out); 

    const unsigned int len = pt.size(0); 
    const unsigned int threads = 1024;   
    const dim3 blk = BLOCKS(threads, len, 3, 1); 
    
    AT_DISPATCH_FLOATING_TYPES(pt.scalar_type(), "PxPyPzK", ([&]
    {
        PxPyPzK<scalar_t><<<blk, threads>>>(
            pt.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
           eta.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
           phi.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
           out.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            len
        );
    })); 

    c10::cuda::set_device(current_device);
    return out;  
}

torch::Tensor _PxPyPzE(torch::Tensor pmu){
    const auto current_device = c10::cuda::current_device();
    c10::cuda::set_device(pmu.get_device()); 

    const torch::TensorOptions op = _MakeOp(pmu); 
    torch::Tensor out = torch::zeros_like(pmu, op);
    CHECK_INPUT(out); CHECK_INPUT(pmu); 

    const unsigned int len = out.size(0); 
    const unsigned int threads = 1024;   
    const dim3 blk = BLOCKS(threads, len, 4, 1); 
    
    AT_DISPATCH_FLOATING_TYPES(out.scalar_type(), "PxPyPz3K", ([&]
    {
        PxPyPzEK<scalar_t><<<blk, threads>>>(
            pmu.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            out.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            len
        );
    })); 
    c10::cuda::set_device(current_device);
    return out;  
}

#endif
