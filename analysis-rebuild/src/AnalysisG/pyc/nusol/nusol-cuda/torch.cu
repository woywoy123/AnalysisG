#include "hip/hip_runtime.h"
#include <operators/operators-cuda.h>
#include <transform/cartesian-cuda.h>
#include <transform/polar-cuda.h>
#include <physics/physics-cuda.h>
#include <torch/torch.h>
#include <stdio.h>
#include <vector>
#include <map>

#ifndef NUSOL_CUDA_KERNEL_H
#define NUSOL_CUDA_KERNEL_H
#include "kernel.cu"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), "#x must be on CUDA")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), "#x must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

const dim3 BLOCKS(
    const unsigned int threads, const unsigned int len, 
    const unsigned int dy, const unsigned int dz)
{
    const dim3 blocks( (len + threads -1) / threads, dy, dz); 
    return blocks; 
}

const dim3 BLOCKS(const unsigned int threads, const unsigned int dx){
    const dim3 blocks( (dx + threads -1) / threads); 
    return blocks; 
}

const dim3 BLOCKS(const unsigned int threads, const unsigned int dx, const unsigned int dy){
    const dim3 blocks( (dx + threads -1) / threads, dy); 
    return blocks; 
}

const torch::TensorOptions _MakeOp(torch::Tensor v1){
    return torch::TensorOptions().dtype(v1.scalar_type()).device(v1.device()); 
}

const std::map<std::string, torch::Tensor> _convert(torch::Tensor met_phi){   
    torch::Tensor met = met_phi.index({torch::indexing::Slice(), 0}); 
    torch::Tensor phi = met_phi.index({torch::indexing::Slice(), 1}); 
    torch::Tensor met_x = transform::cuda::Px(met, phi);
    torch::Tensor met_y = transform::cuda::Py(met, phi);

    std::map<std::string, torch::Tensor> out;
    out["met_xy"] = torch::cat({met_x, met_y}, -1); 
    return out; 
}

std::map<std::string, torch::Tensor> _convert(torch::Tensor pmu1, torch::Tensor pmu2){  
    const unsigned int dim_i = pmu1.size(0); 
    torch::Tensor com = torch::cat({pmu1, pmu2}, 0); 
    com = transform::cuda::PxPyPzE(com);

    std::map<std::string, torch::Tensor> out;
    out["pmc1"] = com.index({torch::indexing::Slice(0, dim_i)}); 
    out["pmc2"] = com.index({torch::indexing::Slice(dim_i, dim_i*2)}); 
    return out; 
}

torch::Tensor _format(std::vector<torch::Tensor> v){  
    std::vector<torch::Tensor> out; 
    for (torch::Tensor i : v){out.push_back(i.view({-1, 1}));}
    return torch::cat(out, -1); 
}

torch::Tensor _Shape_Matrix(torch::Tensor inpt, std::vector<long> vec){
    const unsigned int len_i = inpt.size(0); 
    const unsigned int len_j = vec.size(); 
    const unsigned int threads = 1024; 
    const torch::TensorOptions op = _MakeOp(inpt); 

    torch::Tensor out = torch::zeros_like(inpt); 
    torch::Tensor vecT = torch::zeros({1, 1, len_j}, op).to(torch::kCPU); 
    for (unsigned int i(0); i < len_j; ++i){ vecT[0][0][i] += vec[i]; }
    vecT = vecT.to(op); 

    const dim3 blk = BLOCKS(threads, len_i, len_j, len_j);
    AT_DISPATCH_FLOATING_TYPES(out.scalar_type(), "ShapeMatrix", ([&]{
        _ShapeKernel<scalar_t><<< blk, threads >>>(
                 out.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                vecT.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                len_i, len_j, len_j, false); 
    })); 

    return out; 
} 

unsigned int _getDim(torch::Tensor inpt, int index1, int index2, double step_size){
    torch::Tensor dx;
    dx = inpt.index({torch::indexing::Slice(), index1});
    dx -= inpt.index({torch::indexing::Slice(), index2});
    dx = torch::abs(dx)/step_size; 
    return torch::max(dx).item<int>(); 
}

const torch::Tensor _Expand_Matrix(torch::Tensor inpt, torch::Tensor source){
    const unsigned int threads = 1024; 
    const unsigned int len_i = inpt.size(0);
    const unsigned int len_k = source.size(1); 
    const torch::TensorOptions op = _MakeOp(inpt); 
    source = source.view({source.size(0), len_k, -1}); 
  
    const unsigned int len_j = source.size(2); 
    const dim3 blk = BLOCKS(threads, len_i, len_k, len_j);
    torch::Tensor out = torch::zeros_like(inpt, op); 

    AT_DISPATCH_FLOATING_TYPES(out.scalar_type(), "ShapeMatrix", ([&]{
        _ShapeKernel<scalar_t><<< blk, threads >>>(
                out.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                source.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                len_i, (len_k > inpt.size(1)) ? inpt.size(1) : len_k, len_j, true); 
    })); 

    return out; 
}


std::tuple<torch::Tensor, torch::Tensor> _DotMatrix(torch::Tensor MET_xy, torch::Tensor H, torch::Tensor Shape){
    const unsigned int threads = 1024; 
    const unsigned int dim_i = MET_xy.size(0); 
    const dim3 blk = BLOCKS(threads, dim_i, 3, 3);
    const torch::TensorOptions op = _MakeOp(H); 

    MET_xy = _Expand_Matrix(H, MET_xy); 
    torch::Tensor   X = torch::zeros({dim_i, 3, 3}, op);
    torch::Tensor dNu = torch::zeros({dim_i, 3, 3}, op); 

    AT_DISPATCH_FLOATING_TYPES(MET_xy.scalar_type(), "NuMatrix", ([&]{
        _V0_deltaK<scalar_t><<< blk, threads >>>(
                 X.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
               dNu.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            MET_xy.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
             Shape.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                 H.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                 dim_i); 
    })); 

    return {operators::cuda::Mul(X, dNu), dNu}; 
}

torch::Tensor _Base_Matrix(torch::Tensor pmc_b, torch::Tensor pmc_mu, torch::Tensor masses_W_top_nu){
    const unsigned int threads = 1024; 
    const unsigned int len_i = pmc_b.size(0); 
    const unsigned int len_m = masses_W_top_nu.size(0); 

    torch::Tensor beta2_b   = physics::cuda::Beta2(pmc_b); 
    torch::Tensor mass2_b   = physics::cuda::M2(pmc_b); 
    
    torch::Tensor beta2_mu  = physics::cuda::Beta2(pmc_mu);
    torch::Tensor mass2_mu  = physics::cuda::M2(pmc_mu); 
    
    torch::Tensor costheta  = operators::cuda::CosTheta(pmc_b, pmc_mu, 3);

    // [Z/Om, 0, x1 - p_mu], [ w * Z/Om, 0, y1 ], [0, Z, 0]
    torch::Tensor out = torch::zeros({len_i, 3, 3}, _MakeOp(costheta)); 
    const dim3 blk = BLOCKS(threads, len_i, 3, 3); 
    AT_DISPATCH_FLOATING_TYPES(costheta.scalar_type(), "BaseMatrix", ([&]
    {
        _H_Base<scalar_t><<< blk, threads>>>(
                    out.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

                beta2_b.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                mass2_b.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                  pmc_b.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 

               beta2_mu.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
               mass2_mu.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                 pmc_mu.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 

               costheta.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
        masses_W_top_nu.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
               len_i, len_m); 

        _Base_Matrix_Nan<scalar_t><<< blk, threads>>>(
                out.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                len_i, len_m); 
    })); 
    return out; 
} 

torch::Tensor _Base_Matrix_H(torch::Tensor pmc_b, torch::Tensor pmc_mu, torch::Tensor masses){
    torch::Tensor base  = _Base_Matrix(pmc_b, pmc_mu, masses);
    const torch::TensorOptions op = _MakeOp(base); 
    const unsigned int threads = 1024; 
    const unsigned int dim_i = pmc_b.size(0); 
    const dim3 blk      = BLOCKS(threads, dim_i, 6, 3); 
    const dim3 blk_     = BLOCKS(threads, dim_i, 3, 3); 
    const dim3 blk_dot  = BLOCKS(threads, dim_i, 9, 3);   

    torch::Tensor phi   = transform::cuda::Phi(pmc_mu); 
    torch::Tensor theta = physics::cuda::Theta(pmc_mu); 

    torch::Tensor Rx = _Expand_Matrix(base, pmc_b); 
    torch::Tensor Rx_ = torch::zeros({dim_i, 3, 3, 3}, op); 
    torch::Tensor Rz  = torch::zeros({dim_i, 3, 3}, op); 
    torch::Tensor Ry  = torch::zeros({dim_i, 3, 3}, op); 

    torch::Tensor RzT = torch::zeros({dim_i, 3, 3}, op); 
    torch::Tensor RyT = torch::zeros({dim_i, 3, 3}, op); 
    torch::Tensor RxT = torch::zeros({dim_i, 3, 3}, op);

    AT_DISPATCH_FLOATING_TYPES(phi.scalar_type(), "BaseMatrixH", ([&]{
        _Base_Matrix_H_Kernel<scalar_t><<< blk, threads >>>(
                Ry.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                Rz.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

               RyT.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
               RzT.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

               phi.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
             theta.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
               dim_i); 

        _Rz_Rx_Ry_dot_K<scalar_t><<< blk_dot, threads >>>(
               Rx_.packed_accessor64<scalar_t, 4, torch::RestrictPtrTraits>(), 
                Rx.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                Ry.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                Rz.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                dim_i, 3, 3); 
    })); 
    Rx = Rx_.sum(-1); 

    AT_DISPATCH_FLOATING_TYPES(phi.scalar_type(), "BaseMatrixH", ([&]{
        _Base_Matrix_H_Kernel<scalar_t><<< blk_, threads >>>(
               RxT.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                Rx.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
               dim_i);

        _Rz_Rx_Ry_dot_K<scalar_t><<< blk_dot, threads >>>(
                Rx_.packed_accessor64<scalar_t, 4, torch::RestrictPtrTraits>(), 
                RxT.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                RzT.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                RyT.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                dim_i, 3, 3); 
        
        _dot_K<scalar_t><<< blk_, threads >>>(
                Rx.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                Rx_.packed_accessor64<scalar_t, 4, torch::RestrictPtrTraits>(), 
                base.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                dim_i, 3, 3); 
    })); 
    return Rx; 
}

std::tuple<torch::Tensor, torch::Tensor> _Intersection(torch::Tensor A, torch::Tensor B, const double null){
    const unsigned int dim_i = A.size(0); 
    const unsigned int threads = 1024; 
    const dim3 blk = BLOCKS(threads, dim_i, 3, 3); 
   
    torch::Tensor det_A = operators::cuda::Determinant(A); 
    torch::Tensor det_B = operators::cuda::Determinant(B); 
    
    AT_DISPATCH_FLOATING_TYPES(det_A.scalar_type(), "Swap", ([&]{
        _SwapAB_K<scalar_t><<< blk, threads >>>(
                det_A.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                det_B.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                    A.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                    B.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                dim_i); 
    })); 

    std::tuple<torch::Tensor, torch::Tensor> v = operators::cuda::Inverse(A, true);
    torch::Tensor ignore = std::get<1>(v); 
    torch::Tensor imag = std::get<0>(v); 

    imag = operators::cuda::Mul(imag, B);
    imag = torch::linalg::eigvals(imag); 
    
    const torch::TensorOptions op = _MakeOp(A); 
    const unsigned int dim_eig = imag.size(-1); 
    const dim3 blk_ = BLOCKS(threads, dim_i, 9, dim_eig); 
    std::vector<signed long> dims = {dim_i, dim_eig, 3, 3}; 

    torch::Tensor G   = torch::zeros(dims, op); 
    torch::Tensor L   = torch::zeros(dims, op);
    torch::Tensor O   = torch::zeros(dims, op);
    torch::Tensor swp = torch::zeros({dim_i, dim_eig}, op.dtype(torch::kBool));  

    unsigned int size_swap = sizeof(unsigned int)*18; 
    unsigned int size_det  = sizeof(unsigned int)*12;

    // Oh the joy of C++.....
    unsigned int *sy, *sz, *dy, *dz;

    unsigned int _y[18] = {
        1, 1, 1, 0, 0, 0, 2, 2, 2, 
        0, 0, 0, 1, 1, 1, 2, 2, 2
    };

    unsigned int _z[18] = {
        1, 0, 2, 1, 0, 2, 1, 0, 2, 
        0, 1, 2, 0, 1, 2, 0, 1, 2
    }; 

    unsigned int _dy[12] = {
        1, 1, 2, 2, 
        0, 0, 2, 2, 
        0, 0, 1, 1
    }; 

    unsigned int _dz[12] = {
        1, 2, 1, 2, 
        0, 2, 0, 2, 
        0, 1, 0, 1
    }; 

    uint8_t dev = G.get_device(); 
    hipSetDevice(dev); 
    hipMalloc(&sy, size_swap); 
    hipMalloc(&sz, size_swap); 
    hipMalloc(&dy, size_det); 
    hipMalloc(&dz, size_det); 
    hipMemcpy(sy, &_y , size_swap, hipMemcpyHostToDevice); 
    hipMemcpy(sz, &_z , size_swap, hipMemcpyHostToDevice); 
    hipMemcpy(dy, &_dy, size_det , hipMemcpyHostToDevice); 
    hipMemcpy(dz, &_dz, size_det , hipMemcpyHostToDevice); 

    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(imag.scalar_type(), "imaginary", ([&]
    {
        _imagineK<scalar_t><<< blk_, threads >>>(
            G.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
            A.packed_accessor64<double, 3, torch::RestrictPtrTraits>(), 
            B.packed_accessor64<double, 3, torch::RestrictPtrTraits>(), 
         imag.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            dim_eig, dim_i); 
        
        _degenerateK<scalar_t><<< blk_, threads >>>(
            L.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
          swp.packed_accessor32<bool  , 2, torch::RestrictPtrTraits>(), 
            G.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
            dim_eig, dim_i, sy, sz); 

        _CoFactorK<scalar_t><<< blk_, threads >>>(
            G.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
            L.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
            dim_eig, dim_i, dy, dz);

        _FactorizeK<scalar_t><<< blk_, threads >>>(
            O.packed_accessor64<double, 4, torch::RestrictPtrTraits>(),  
            L.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
            G.packed_accessor64<double, 4, torch::RestrictPtrTraits>(),
            dim_eig, dim_i, null); 

        _SwapXY_K<scalar_t><<< blk_, threads >>>(
            G.packed_accessor64<double, 4, torch::RestrictPtrTraits>(),
            O.packed_accessor64<double, 4, torch::RestrictPtrTraits>(),  
          swp.packed_accessor32<bool  , 2, torch::RestrictPtrTraits>(), 
            dim_eig, dim_i); 
    }));  

    imag = torch::linalg_cross(G.view({-1, dim_eig*dim_eig, 1, 3}), A.view({-1, 1, 3, 3})); 
    imag = torch::transpose(imag, 2, 3);
    imag = std::get<1>(torch::linalg::eig(imag));
    imag = torch::transpose(imag, 2, 3).view({dim_i, -1, 3, 3}).contiguous();

    dims = {dim_i, dim_eig*2, 3, 3}; 
    swp = torch::zeros(dims, op);  
    O   = torch::zeros(dims, op);
    L   = torch::zeros(dims, op);

    const dim3 blk__ = BLOCKS(threads, dim_i, dims[1]*3, 3); 
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(imag.scalar_type(), "intersections", ([&]
    {
        _intersectionK<scalar_t><<< blk__, threads >>>(
            O.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
            L.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
          swp.packed_accessor64<double, 4, torch::RestrictPtrTraits>(),

            A.packed_accessor64<double, 3, torch::RestrictPtrTraits>(), 
            G.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
         imag.packed_accessor64<scalar_t, 4, torch::RestrictPtrTraits>(),
            dim_i, dims[1]); 
    })); 

    torch::Tensor diag = torch::pow(O.sum({-1}), 2) + torch::pow(L.sum({-1}), 2); 
    torch::Tensor id   = std::get<1>(torch::log(diag).sort(-1, false)); 
    torch::Tensor diag_sol = torch::zeros({dim_i, dim_eig*2, 3   }, op); 
    torch::Tensor sols_vec = torch::zeros({dim_i, dim_eig*2, 3, 3}, op); 

    const dim3 blk_r = BLOCKS(threads, dim_i, dim_eig*2, 9);  
    AT_DISPATCH_FLOATING_TYPES(diag_sol.scalar_type(), "sols", ([&]
    {
        _SolsK<scalar_t><<< blk_r, threads >>>(
            diag_sol.packed_accessor64<double, 3, torch::RestrictPtrTraits>(), 
            sols_vec.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
            
                  id.packed_accessor32<long, 3, torch::RestrictPtrTraits>(),  
                diag.packed_accessor64<double, 3, torch::RestrictPtrTraits>(), 
                 swp.packed_accessor64<double, 4, torch::RestrictPtrTraits>(), 
              ignore.packed_accessor64<bool, 2, torch::RestrictPtrTraits>(), 
                dim_i, dim_eig, null);  
    })); 

    hipFree(sy); 
    hipFree(sz); 
    hipFree(dy); 
    hipFree(dz); 

    return {sols_vec, diag_sol};
}

std::map<std::string, torch::Tensor> _Nu(
        torch::Tensor pmc_b, torch::Tensor pmc_mu, 
        torch::Tensor met_xy, torch::Tensor masses, 
        torch::Tensor sigma)
{
    torch::Tensor H = _Base_Matrix_H(pmc_b, pmc_mu, masses); 
    torch::Tensor shape = _Shape_Matrix(H, {0, 0, 1});
    sigma = _Expand_Matrix(H, sigma.view({-1, 2, 2})) + shape; 
    sigma = operators::cuda::Inverse(sigma) - shape;
    torch::Tensor X = std::get<0>(_DotMatrix(met_xy, H, sigma)); 
 
    const unsigned int dim_i = sigma.size(0); 
    const unsigned int threads = 1024; 
    const dim3 blk = BLOCKS(threads, dim_i, 3, 3);
    AT_DISPATCH_FLOATING_TYPES(sigma.scalar_type(), "derivative", ([&]
    {
        _DerivativeK<scalar_t><<< blk, threads >>>(
            sigma.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            X.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            dim_i); 
    })); 

    sigma = operators::cuda::Mul(X, sigma); 
    AT_DISPATCH_FLOATING_TYPES(sigma.scalar_type(), "derivative", ([&]
    {
        _transSumK<scalar_t><<< blk, threads >>>(
            shape.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            sigma.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            dim_i); 
    })); 

    std::map<std::string, torch::Tensor> output;
    output["M"] = shape; 
    output["H"] = H;
    output["X"] = X; 
    return output; 
}

std::map<std::string, torch::Tensor> _Nu(
        torch::Tensor pmc_b, torch::Tensor pmc_mu, 
        torch::Tensor met_xy, torch::Tensor masses, 
        torch::Tensor sigma, const double null)
{
    std::map<std::string, torch::Tensor> nu; 
    nu = _Nu(pmc_b, pmc_mu, met_xy, masses, sigma); 

    torch::Tensor M = nu["M"]; 
    torch::Tensor H = nu["H"]; 
    torch::Tensor X = nu["X"]; 

    std::tuple<torch::Tensor, torch::Tensor> sols; 
    sols = _Intersection(M, _Shape_Matrix(M, {1, 1, -1}), null); 

    torch::Tensor sec = std::get<0>(sols);  
    const torch::TensorOptions op = _MakeOp(sec); 
    const unsigned int threads = 1024; 
    const unsigned int dim_i = sec.size(0);
    const unsigned int dim_eig = sec.size(1);  
    const unsigned int dim_j = 3; 

    std::vector<signed long> dims = {dim_i, dim_eig*3, dim_j}; 
    torch::Tensor sol_chi2 = torch::zeros(dims, op); 
    torch::Tensor sol_vecs = torch::zeros(dims, op); 

    const dim3 blk = BLOCKS(threads, dim_i, dim_eig, dim_j*dim_j); 
    AT_DISPATCH_FLOATING_TYPES(X.scalar_type(), "chi2", ([&]
    {
        _Y_dot_X_dot_Y<scalar_t><<< blk, threads >>>(
            sol_chi2.packed_accessor64<double, 3, torch::RestrictPtrTraits>(), 
            sol_vecs.packed_accessor64<double, 3, torch::RestrictPtrTraits>(), 

                   X.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                 sec.packed_accessor64<double  , 4, torch::RestrictPtrTraits>(), 
            dim_i, dim_eig, dim_j); 
    })); 

    sol_chi2 = sol_chi2.sum(-1);
    torch::Tensor id = std::get<1>(sol_chi2.sort(-1, false)); 
    
    torch::Tensor _nu_v = torch::zeros(dims, op); 
    torch::Tensor _chi2 = torch::zeros_like(sol_chi2, op); 

    AT_DISPATCH_FLOATING_TYPES(H.scalar_type(), "Nu", ([&]
    {
        _NuK<scalar_t><<< blk, threads >>>(
          _nu_v.packed_accessor64<double, 3, torch::RestrictPtrTraits>(), 
          _chi2.packed_accessor64<double, 2, torch::RestrictPtrTraits>(), 

             id.packed_accessor32<    long, 2, torch::RestrictPtrTraits>(), 
              H.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

         sol_vecs.packed_accessor64<double, 3, torch::RestrictPtrTraits>(), 
         sol_chi2.packed_accessor64<double, 2, torch::RestrictPtrTraits>(), 
            dim_i, dim_eig, dim_j*dim_j); 
    })); 

    std::map<std::string, torch::Tensor> output; 
    int max_len = torch::max((_chi2 >= 0).sum(-1)).item<int>(); 
    output["NuVec"] = _nu_v.index({
            torch::indexing::Slice(), 
            torch::indexing::Slice(dims[1] - max_len, torch::indexing::None),
            torch::indexing::Slice()
    }); 
 
    output["chi2"]  = _chi2.index({
            torch::indexing::Slice(), 
            torch::indexing::Slice(dims[1] - max_len, torch::indexing::None)
    }); 

    return output;  
}

std::map<std::string, torch::Tensor> _NuNu(
        torch::Tensor pmc_b1, torch::Tensor pmc_b2, torch::Tensor pmc_l1, torch::Tensor pmc_l2,
        torch::Tensor masses1, torch::Tensor masses2, torch::Tensor met_xy, const double null
){
    std::tuple<torch::Tensor, torch::Tensor> X;
    std::map<std::string, torch::Tensor> output; 
    const torch::TensorOptions op = _MakeOp(masses1); 
    const unsigned int threads = 1024;
    const unsigned int dim_j = 3;  
    const unsigned int dim_i = pmc_b1.size(0); 
    const dim3 blk   = BLOCKS(threads, dim_i, dim_j, 2); 
    const dim3 blk_  = BLOCKS(threads, dim_i,    27, 2);   
    const dim3 blk_d = BLOCKS(threads, dim_i,     9, 2); 

    torch::Tensor H1 = _Base_Matrix_H(pmc_b1, pmc_l1, masses1); 
    torch::Tensor H2 = _Base_Matrix_H(pmc_b2, pmc_l2, masses2); 

    torch::Tensor circl = _Shape_Matrix(H1, {1, 1, -1}); 
    torch::Tensor H_perp_1 = H1.clone(); 
    torch::Tensor H_perp_2 = H2.clone(); 

    torch::Tensor N1 = torch::zeros({dim_i, dim_j, dim_j, dim_j}, op); 
    torch::Tensor N2 = torch::zeros({dim_i, dim_j, dim_j, dim_j}, op); 

    AT_DISPATCH_FLOATING_TYPES(H1.scalar_type(), "H_perp", ([&]{
        _H_perp_K<scalar_t><<< blk, threads >>>(
            H_perp_1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            H_perp_2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            dim_i, dim_j, 2); 
    })); 

    torch::Tensor H_inv_1 = operators::cuda::Inverse(H_perp_1); 
    torch::Tensor H_inv_2 = operators::cuda::Inverse(H_perp_2);

    AT_DISPATCH_FLOATING_TYPES(H1.scalar_type(), "YT_DOT_X_DOTY", ([&]{
        _YT_dot_X_dot_Y<scalar_t><<< blk_, threads >>>(
            N1.packed_accessor64<scalar_t, 4, torch::RestrictPtrTraits>(), 
            N2.packed_accessor64<scalar_t, 4, torch::RestrictPtrTraits>(), 

       H_inv_1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
         circl.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

       H_inv_2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
         circl.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            dim_i, dim_j, 2); 
    })); 
    
    N1 = N1.sum(-1); 
    N2 = N2.sum(-1); 
     
    X = _DotMatrix(met_xy, circl, N2); 
    torch::Tensor n_ = std::get<0>(X); 
    torch::Tensor S  = std::get<1>(X); 
    
    X = _Intersection(N1, n_, null);
    const unsigned int dim_eig = std::get<0>(X).size(1);
    std::vector<signed long> dims = {dim_i, dim_eig*3, dim_j}; 

    torch::Tensor sol  = std::get<0>(X); 
    torch::Tensor diag = std::get<1>(X).view({dim_i, -1});
    torch::Tensor id   = std::get<1>(torch::log(diag).sort(-1, false)); 
    
    torch::Tensor v    = torch::zeros(dims, op);
    torch::Tensor v_   = torch::zeros(dims, op);

    torch::Tensor nu   = torch::zeros(dims, op); 
    torch::Tensor nu_  = torch::zeros(dims, op); 
    torch::Tensor dnu  = torch::zeros({dim_i, dim_eig*3}, op); 

    torch::Tensor K1   = torch::zeros({dim_i, dim_j, dim_j}, op);
    torch::Tensor K2   = torch::zeros({dim_i, dim_j, dim_j}, op);
   
    const dim3 blk__ = BLOCKS(threads, dim_i, dims[1], 6);   
    AT_DISPATCH_FLOATING_TYPES(N1.scalar_type(), "DOTS", ([&]
    {
        _DotK<scalar_t><<< blk__, threads >>>(
             v.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            v_.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

             S.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),  
           sol.packed_accessor64<scalar_t, 4, torch::RestrictPtrTraits>(), 
            dim_i, dims[1], 6); 

        _K_Kern<scalar_t><<< blk_d, threads >>>(
            K1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            K2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

            H1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
       H_inv_1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

            H2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
       H_inv_2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            dim_i, 9, 2); 

        _NuNuK<scalar_t><<< blk__, threads >>>(
            nu.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
           nu_.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
           dnu.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 

            K1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            K2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

             v.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
            v_.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

            id.packed_accessor32<long    , 2, torch::RestrictPtrTraits>(), 
          diag.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            dim_i, dims[1], 6); 
    })); 
   
    torch::Tensor none = (nu.sum(-1) != 0).sum(-1);  
    int max_len = torch::max(none).item<int>(); 
    if (!max_len){max_len = 1;}
    output["NuVec_1"] = nu.index({
            torch::indexing::Slice(), 
            torch::indexing::Slice(0, max_len), 
            torch::indexing::Slice()
    }); 

    output["NuVec_2"] = nu_.index({
            torch::indexing::Slice(), 
            torch::indexing::Slice(0, max_len), 
            torch::indexing::Slice()
    }); 

    output["diagonal"] = dnu.index({
            torch::indexing::Slice(), 
            torch::indexing::Slice(0, max_len)
    });   
    
    output["n_"] = n_; 
    output["H_perp_1"] = H_perp_1; 
    output["H_perp_2"] = H_perp_2;
    output["NoSols"] = none == 0;   
    return output; 
}



void _MassMatrix(
        const double mtop_l, const double mtop_u, 
        const double mw_l, const double mw_u, 
        torch::Tensor masses, const float steps)
{
    const float step_t = (mtop_u - mtop_l)/steps; 
    const float step_w = (mw_u - mw_l)/steps; 
    const unsigned int threads = 1024; 
    const dim3 blk = BLOCKS(threads, (int)steps, 2); 

    AT_DISPATCH_ALL_TYPES(masses.scalar_type(), "masses", ([&]{
        _MassKernel<scalar_t><<< blk, threads >>>(
                masses.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                mtop_l, step_t, mw_l, step_w, (int)steps
        ); 
    })); 
}

torch::Tensor _create_event_nlep(torch::Tensor edge_idx, torch::Tensor pid)
{
    const unsigned int threads = 1024; 
    const unsigned int nodes = pid.size(0); 
    torch::Tensor maps = torch::zeros({nodes, nodes}, _MakeOp(pid)); 
    const dim3 blk = BLOCKS(threads, nodes, nodes);  
    AT_DISPATCH_ALL_TYPES(edge_idx.scalar_type(), "n-leps", ([&]{
        _lep_map<scalar_t><<< blk, threads >>>(
                maps.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                 pid.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            edge_idx.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                nodes
        ); 
    })); 
    maps = maps.sum({-1}, true); 
    return torch::cat({pid, maps}, -1); 
}

std::vector<torch::Tensor> _create_mapping(torch::Tensor edge_idx, torch::Tensor pid, torch::Tensor batch)
{
    const unsigned int threads = 1024; 
    const unsigned int idx_s = edge_idx.size(1);
    const dim3 blk = BLOCKS(threads, idx_s, idx_s);
    torch::Tensor llbb_nu = torch::zeros({idx_s*idx_s, 6}, _MakeOp(pid)); 
    batch = batch.to(_MakeOp(pid)); 

    AT_DISPATCH_ALL_TYPES(pid.scalar_type(), "mapping", ([&]{
        _MappingKernel<scalar_t><<<blk, threads>>>(
                 llbb_nu.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                   batch.packed_accessor64<scalar_t, 1, torch::RestrictPtrTraits>(), 
                edge_idx.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                     pid.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                    idx_s
        ); 
    })); 

    torch::Tensor msk_nu    = llbb_nu.index({torch::indexing::Slice(), 4}) >= 1; 
    torch::Tensor msk_nunu  = llbb_nu.index({torch::indexing::Slice(), 5}) >= 1; 
    torch::Tensor pair_nu   = llbb_nu.index({msk_nu}); 
    torch::Tensor pair_nunu = llbb_nu.index({msk_nunu}); 
    return {pair_nu, pair_nunu}; 
}

std::vector<torch::Tensor> _viable_solutions(
        torch::Tensor pairs, torch::Tensor mass_matrix, 
        torch::Tensor pmc, torch::Tensor t_met_xy, torch::Tensor batch,
        const double null)
{
    const unsigned int threads = 1024;
    const unsigned int lx = pairs.size(0); 
    const unsigned int ly = mass_matrix.size(0); 
    const unsigned int lz = 4*4 + 3;  
    const unsigned int len_i = ly*ly; 
    torch::Tensor met_xy = t_met_xy.index({batch});  

    torch::Tensor pmc_b1 = torch::zeros({lx, len_i, 4}, _MakeOp(pmc));
    torch::Tensor pmc_b2 = torch::zeros({lx, len_i, 4}, _MakeOp(pmc));

    torch::Tensor pmc_l1 = torch::zeros({lx, len_i, 4}, _MakeOp(pmc));
    torch::Tensor pmc_l2 = torch::zeros({lx, len_i, 4}, _MakeOp(pmc));
    torch::Tensor pair_i = torch::zeros({lx, len_i, 4}, _MakeOp(pairs)); 

    torch::Tensor mass_m1 = torch::zeros({lx, len_i, 3}, _MakeOp(pmc));
    torch::Tensor mass_m2 = torch::zeros({lx, len_i, 3}, _MakeOp(pmc));
    torch::Tensor met_xy_ = torch::zeros({lx, len_i, 2}, _MakeOp(pmc)); 

    const dim3 blk = BLOCKS(threads, lx, len_i, lz); 
    AT_DISPATCH_ALL_TYPES(pmc.scalar_type(), "assigns", ([&]{
        _assignment_kernel<scalar_t><<<blk, threads>>>(
                 pmc_b1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                 pmc_b2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

                 pmc_l1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                 pmc_l2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

                 pair_i.packed_accessor64<long    , 3, torch::RestrictPtrTraits>(),
                mass_m1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                mass_m2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                met_xy_.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 

                 met_xy.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
            mass_matrix.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                    pmc.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                  pairs.packed_accessor64<long    , 2, torch::RestrictPtrTraits>(),
                    lx, ly, lz
        );
    })); 

    torch::Tensor dia_sol_o = torch::ones({lx, len_i}, _MakeOp(mass_m1))*-1; 
    torch::Tensor nu_1 = torch::zeros({lx, len_i, 4}, _MakeOp(mass_m1)); 
    torch::Tensor nu_2 = torch::zeros({lx, len_i, 4}, _MakeOp(mass_m2)); 

    const dim3 blk_ = BLOCKS(threads, ly, ly, 5); 
    for (unsigned int x(0); x < lx; ++x){ 
        std::map<std::string, torch::Tensor> nus;
        nus = _NuNu(pmc_b1[x], pmc_b2[x], pmc_l1[x], pmc_l2[x], mass_m1[x], mass_m2[x], met_xy_[x], null); 
        torch::Tensor nu1 = nus["NuVec_1"]; 
        torch::Tensor nu2 = nus["NuVec_2"]; 
        torch::Tensor dia = nus["diagonal"]; 
        torch::Tensor noS = nus["NoSols"]; 
        AT_DISPATCH_ALL_TYPES(pmc.scalar_type(), "builder", ([&]{
            _builder_nunu<scalar_t><<<blk_, threads>>>(
               dia_sol_o.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
                    nu_1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(), 
                    nu_2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),

                    noS.packed_accessor64<bool    , 1, torch::RestrictPtrTraits>(),
                    dia.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    nu1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    nu2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    x, ly, 5); 
        })); 
    }

    torch::Tensor dia_min_comb = -1*torch::ones({t_met_xy.size(0), len_i}, _MakeOp(dia_sol_o));  
    torch::Tensor dia_min_mass = -1*torch::ones({t_met_xy.size(0), lx   }, _MakeOp(dia_sol_o)); 
    const dim3 blk_x = BLOCKS(threads, lx*len_i); 
    AT_DISPATCH_ALL_TYPES(pmc.scalar_type(), "finder", ([&]{
        _min_finder<scalar_t><<<blk_x, threads>>>(
                dia_min_mass.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                dia_min_comb.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                      pair_i.packed_accessor64<long    , 3, torch::RestrictPtrTraits>(),
                       batch.packed_accessor64<long    , 1, torch::RestrictPtrTraits>(),
                   dia_sol_o.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    lx, len_i);  
    })); 

    torch::Tensor optimal = torch::cat({dia_min_comb, dia_min_mass}, -1); 
    optimal.index_put_({optimal < 0}, 1); 

    torch::Tensor min_ = std::get<0>(optimal.min({-1})); 
    torch::Tensor nu_1f = torch::zeros({t_met_xy.size(0), 4}, _MakeOp(pmc)); 
    torch::Tensor ms_1f = torch::zeros({t_met_xy.size(0), 3}, _MakeOp(pmc)); 

    torch::Tensor nu_2f = torch::zeros({t_met_xy.size(0), 4}, _MakeOp(pmc)); 
    torch::Tensor ms_2f = torch::zeros({t_met_xy.size(0), 3}, _MakeOp(pmc)); 
  
    torch::Tensor combi = torch::zeros({t_met_xy.size(0), 4}, _MakeOp(pair_i)); 

    const dim3 blk_o = BLOCKS(threads, lx, len_i, 4); 
    AT_DISPATCH_ALL_TYPES(pmc.scalar_type(), "populate", ([&]{
        _populate<scalar_t><<<blk_o, threads>>>(
            nu_1f.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
            ms_1f.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
            nu_2f.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
            ms_2f.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
            combi.packed_accessor64<long    , 2, torch::RestrictPtrTraits>(),

            nu_1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
         mass_m1.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
            nu_2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
         mass_m2.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
          pair_i.packed_accessor64<long    , 3, torch::RestrictPtrTraits>(),

       dia_sol_o.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(), 
            min_.packed_accessor64<scalar_t, 1, torch::RestrictPtrTraits>(),
           batch.packed_accessor64<long    , 1, torch::RestrictPtrTraits>(),
           lx, ly, 4); 
    })); 
    return {nu_1f, nu_2f, ms_1f, ms_2f, combi, min_.view({-1, 1})}; 
}




// -------------------- Interfaces ------------------------- //
std::map<std::string, torch::Tensor> _NuPolar(
        torch::Tensor pmu_b, torch::Tensor pmu_mu, 
        torch::Tensor met_phi, torch::Tensor masses, 
        torch::Tensor sigma, const double null)
{
    std::map<std::string, torch::Tensor> met = _convert(met_phi); 
    std::map<std::string, torch::Tensor> pmc = _convert(pmu_b, pmu_mu); 

    return _Nu(pmc["pmc1"], pmc["pmc2"], met["met_xy"], masses, sigma, null);
}

std::map<std::string, torch::Tensor> _NuPolar(
        torch::Tensor pt_b, torch::Tensor eta_b, torch::Tensor phi_b, torch::Tensor e_b, 
        torch::Tensor pt_mu, torch::Tensor eta_mu, torch::Tensor phi_mu, torch::Tensor e_mu, 
        torch::Tensor met, torch::Tensor phi, torch::Tensor masses, 
        torch::Tensor sigma, const double null)
{
    torch::Tensor pmu_b = _format({pt_b, eta_b, phi_b, e_b}); 
    torch::Tensor pmu_mu = _format({pt_mu, eta_mu, phi_mu, e_mu}); 
    torch::Tensor met_ = _format({met, phi}); 
    
    std::map<std::string, torch::Tensor> pmc = _convert(pmu_b, pmu_mu); 
    std::map<std::string, torch::Tensor> _met = _convert(met_);
    return _Nu(pmc["pmc1"], pmc["pmc2"], _met["met_xy"], masses, sigma, null);
}


std::map<std::string, torch::Tensor> _NuCart(
        torch::Tensor pmc_b, torch::Tensor pmc_mu, 
        torch::Tensor met_xy, torch::Tensor masses, 
        torch::Tensor sigma, const double null)
{
    return _Nu(pmc_b, pmc_mu, met_xy, masses, sigma, null);
}

std::map<std::string, torch::Tensor> _NuCart(
        torch::Tensor px_b, torch::Tensor py_b, torch::Tensor pz_b, torch::Tensor e_b, 
        torch::Tensor px_mu, torch::Tensor py_mu, torch::Tensor pz_mu, torch::Tensor e_mu, 
        torch::Tensor metx, torch::Tensor mety, torch::Tensor masses, 
        torch::Tensor sigma, const double null)
{
    torch::Tensor pmc_b = _format({px_b, py_b, pz_b, e_b}); 
    torch::Tensor pmc_mu = _format({px_mu, py_mu, pz_mu, e_mu}); 
    torch::Tensor met_ = _format({metx, mety}); 
    
    return _Nu(pmc_b, pmc_mu, met_, masses, sigma, null);
}


std::map<std::string, torch::Tensor> _NuNu(
                torch::Tensor pmc_b1, torch::Tensor pmc_b2, 
                torch::Tensor pmc_l1, torch::Tensor pmc_l2,
                torch::Tensor met_xy, torch::Tensor masses, 
                const double null)
{
    return _NuNu(pmc_b1, pmc_b2, pmc_l1, pmc_l2, masses, masses, met_xy, null); 
}



std::map<std::string, torch::Tensor> _NuNuPolar(
        torch::Tensor pmu_b1 , torch::Tensor pmu_b2, 
        torch::Tensor pmu_mu1, torch::Tensor pmu_mu2, 
        torch::Tensor met_phi, torch::Tensor masses, 
        const double null)
{
    std::map<std::string, torch::Tensor> met = _convert(met_phi); 
    std::map<std::string, torch::Tensor> pmc_b  = _convert(pmu_b1, pmu_b2); 
    std::map<std::string, torch::Tensor> pmc_mu = _convert(pmu_mu1, pmu_mu2); 

    return _NuNu(pmc_b["pmc1"], pmc_b["pmc2"], pmc_mu["pmc1"], pmc_mu["pmc2"], met["met_xy"], masses, null);
}

std::map<std::string, torch::Tensor> _NuNuPolar(
        torch::Tensor pt_b1, torch::Tensor eta_b1, torch::Tensor phi_b1, torch::Tensor e_b1, 
        torch::Tensor pt_b2, torch::Tensor eta_b2, torch::Tensor phi_b2, torch::Tensor e_b2, 

        torch::Tensor pt_mu1, torch::Tensor eta_mu1, torch::Tensor phi_mu1, torch::Tensor e_mu1, 
        torch::Tensor pt_mu2, torch::Tensor eta_mu2, torch::Tensor phi_mu2, torch::Tensor e_mu2, 

        torch::Tensor met, torch::Tensor phi, 
        torch::Tensor masses, const double null)
{
    torch::Tensor pmu_b1 = _format({pt_b1, eta_b1, phi_b1, e_b1});
    torch::Tensor pmu_b2 = _format({pt_b2, eta_b2, phi_b2, e_b2});

    torch::Tensor pmu_mu1 = _format({pt_mu1, eta_mu1, phi_mu1, e_mu1});
    torch::Tensor pmu_mu2 = _format({pt_mu2, eta_mu2, phi_mu2, e_mu2});

    std::map<std::string, torch::Tensor> _met   = _convert(_format({met, phi}));  
    std::map<std::string, torch::Tensor> pmc_b  = _convert(pmu_b1 , pmu_b2); 
    std::map<std::string, torch::Tensor> pmc_mu = _convert(pmu_mu1, pmu_mu2); 

    return _NuNu(pmc_b["pmc1"], pmc_b["pmc2"], pmc_mu["pmc1"], pmc_mu["pmc2"], _met["met_xy"], masses, null);
}

std::map<std::string, torch::Tensor> _NuNuCart(
        torch::Tensor pmc_b1, torch::Tensor pmc_b2, 
        torch::Tensor pmc_mu1, torch::Tensor pmc_mu2,
        torch::Tensor met_xy, torch::Tensor masses, const double null)
{
    return _NuNu(pmc_b1, pmc_b2, pmc_mu1, pmc_mu2, met_xy, masses, null);
}

std::map<std::string, torch::Tensor> _NuNuCart(
        torch::Tensor px_b1, torch::Tensor py_b1, torch::Tensor pz_b1, torch::Tensor e_b1, 
        torch::Tensor px_b2, torch::Tensor py_b2, torch::Tensor pz_b2, torch::Tensor e_b2, 

        torch::Tensor px_mu1, torch::Tensor py_mu1, torch::Tensor pz_mu1, torch::Tensor e_mu1, 
        torch::Tensor px_mu2, torch::Tensor py_mu2, torch::Tensor pz_mu2, torch::Tensor e_mu2, 

        torch::Tensor metx, torch::Tensor mety, torch::Tensor masses, const double null)
{
    torch::Tensor pmc_b1  = _format({px_b1, py_b1, pz_b1, e_b1}); 
    torch::Tensor pmc_b2  = _format({px_b2, py_b2, pz_b2, e_b2}); 
    torch::Tensor pmc_mu1 = _format({px_mu1, py_mu1, pz_mu1, e_mu1}); 
    torch::Tensor pmc_mu2 = _format({px_mu2, py_mu2, pz_mu2, e_mu2}); 
    torch::Tensor met_    = _format({metx, mety}); 
    
    return _NuNu(pmc_b1, pmc_b2, pmc_mu1, pmc_mu2, met_, masses, null);
}


std::map<std::string, torch::Tensor> _CombinatorialCartesian(
        torch::Tensor edge_index, torch::Tensor batch, 
        torch::Tensor pmc, torch::Tensor pid, torch::Tensor met_xy, 
        const double mass_top_l, const double mass_top_u, const double mass_w_l, const double mass_w_u, 
        const double mass_nu, const double null)
{
    int steps = 50;
    torch::Tensor mass_matrix = torch::zeros({steps, 3}, _MakeOp(met_xy)); 
    _MassMatrix(mass_top_l, mass_top_u, mass_w_l, mass_w_u, mass_matrix, steps); 

    pid = pid.to(_MakeOp(edge_index)); 
    pid = _create_event_nlep(edge_index, pid); 
    std::vector<torch::Tensor> nus = _create_mapping(edge_index, pid, batch); 
    torch::Tensor pair_nu   = nus[0]; 
    torch::Tensor pair_nunu = nus[1]; 

    std::vector<torch::Tensor> res; 
    if (!pair_nunu.size(0)){
        torch::Tensor nu_null = torch::cat({torch::zeros_like(met_xy), torch::zeros_like(met_xy)}, -1); 
        res = {nu_null, nu_null, nu_null, nu_null, nu_null, nu_null}; 
    }
    else {res = _viable_solutions(pair_nunu, mass_matrix, pmc, met_xy, batch, null);}

    return {
        {"nu_1f", res[0]}, {"nu_2f", res[1]}, 
        {"ms_1f", res[2]}, {"ms_2f", res[3]}, 
        {"combi", res[4]}, {"min"  , res[5]}
    }; 
}

#endif
